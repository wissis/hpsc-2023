
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

//const int n = 150;
const int n = 50;
//const int range = 10;
const int range = 5;

__device__ __managed__ int bucket[range];
__device__ __managed__ int end = 0;

__global__ void bucket_add(int *bucket, int *key){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index >= n) return;
  int buffer = key[index];
  atomicAdd(&bucket[buffer], 1);
}

__global__ void bucket_remove(int *bucket, int *key, int i, int end){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index >= n + end) return;
  int bucketindex = i;
//  printf("%d %d\n", bucketindex, index);
  key[index + end] = bucketindex;
//  if(index == 0) printf("key: %d\n", key[index+end]);
}

int main() {
//  int n = 50;
//  int range = 5;
//  std::vector<int> key(n);
  int *key;
  hipMallocManaged(&key, n*sizeof(int));
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");

//  const int N = 128;
  const int M = 32;
//  std::vector<int> bucket(range);
  int *bucket;
  hipMallocManaged(&bucket, range*sizeof(int));
  bucket_add<<<(n+M-1)/M, M>>>(bucket, key);
  hipDeviceSynchronize();
//  for (int i=0; i<range; i++) printf("%d ", bucket[i]);
  for (int i=0; i<range; i++){
//    printf("LOOP %d \n", i);
    bucket_remove<<<(n+M-1)/M, M>>>(bucket, key, i, end);
    end = end + bucket[i];
    hipDeviceSynchronize();
  }
 // cudaDeviceSynchronize();

/*
  std::vector<int> bucket(range); 
  for (int i=0; i<range; i++) {
    bucket[i] = 0;
  }
  for (int i=0; i<n; i++) {
    bucket[key[i]]++;
  }
  for (int i=0, j=0; i<range; i++) {
    for (; bucket[i]>0; bucket[i]--) {
      key[j++] = i;
    }
  }
*/
  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");
  hipFree(bucket);
  hipFree(key);
}
